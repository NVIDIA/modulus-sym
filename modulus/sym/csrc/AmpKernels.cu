#include "hip/hip_runtime.h"
/*
From PyTorch:

Copyright (c) 2016-     Facebook, Inc            (Adam Paszke)
Copyright (c) 2014-     Facebook, Inc            (Soumith Chintala)
Copyright (c) 2011-2014 Idiap Research Institute (Ronan Collobert)
Copyright (c) 2012-2014 Deepmind Technologies    (Koray Kavukcuoglu)
Copyright (c) 2011-2012 NEC Laboratories America (Koray Kavukcuoglu)
Copyright (c) 2011-2013 NYU                      (Clement Farabet)
Copyright (c) 2006-2010 NEC Laboratories America (Ronan Collobert, Leon Bottou, Iain Melvin, Jason Weston)
Copyright (c) 2006      Idiap Research Institute (Samy Bengio)
Copyright (c) 2001-2004 Idiap Research Institute (Ronan Collobert, Samy Bengio, Johnny Mariethoz)

From Caffe2:

Copyright (c) 2016-present, Facebook Inc. All rights reserved.

All contributions by Facebook:
Copyright (c) 2016 Facebook Inc.

All contributions by Google:
Copyright (c) 2015 Google Inc.
All rights reserved.

All contributions by Yangqing Jia:
Copyright (c) 2015 Yangqing Jia
All rights reserved.

All contributions by Kakao Brain:
Copyright 2019-2020 Kakao Brain

All contributions from Caffe:
Copyright(c) 2013, 2014, 2015, the respective contributors
All rights reserved.

All other contributions:
Copyright(c) 2015, 2016 the respective contributors
All rights reserved.

Caffe2 uses a copyright model similar to Caffe: each contributor holds
copyright over their contributions to Caffe2. The project versioning records
all such contribution and copyright details. If a contributor wants to further
mark their specific copyright on a particular contribution, they should
indicate their copyright solely in the commit message of the change when it is
committed.

All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.

3. Neither the names of Facebook, Deepmind Technologies, NYU, NEC Laboratories America
   and IDIAP Research Institute nor the names of its contributors may be
   used to endorse or promote products derived from this software without
   specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.
*/
// Modified from https://github.com/pytorch/pytorch/blob/release/1.11/aten/src/ATen/native/cuda/AmpKernels.cu#L181

#include <torch/python.h>
#include <c10/cuda/CUDAStream.h>

using torch::Tensor;

// amp_update_scale_cuda_kernel is launched with a single thread to compute the new scale.
// The scale factor is maintained and updated on the GPU to avoid synchronization.
__global__ void amp_update_scale_cuda_kernel(float* current_scale,
                                             int* growth_tracker,
                                             float* found_inf,
                                             float growth_factor,
                                             float backoff_factor,
                                             int growth_interval,
                                             float max_scale,
                                             float recover_threshold,
                                             int recover_growth_interval) {
  if (*found_inf) {
    *current_scale = (*current_scale)*backoff_factor;
    *growth_tracker = 0;
  } else {
    // Entering this branch means we just carried out a successful step,
    // so growth_tracker is incremented before comparing to growth_interval.
    auto successful = (*growth_tracker) + 1;
    // decide whether to use the recover_growth_interval
    growth_interval = (*current_scale) <= recover_threshold ? recover_growth_interval : growth_interval;
    if (successful == growth_interval) {
      // grow the scale then clamp with max_scale
      *current_scale = min((*current_scale) * growth_factor, max_scale);
      *growth_tracker = 0;
    } else {
      *growth_tracker = successful;
    }
  }
}

// _amp_update_scale_cuda asynchronously updates the scale tensor in place.
//
// Args:
// current_scale:  A one-element cuda float tensor containing the scale value.
// growth_tracker:  A one-element torch.cuda.IntTensor containing the number of recent consecutive unskipped steps.
// found_inf:  A one-element cuda float tensor. If > 0, indicates that infs/nans were found by the relevant
//             prior _amp_non_finite_check_and_unscale_cuda call, and 0 if no infs/nans were found.
// growth_factor:  Multiplier if no infs/NaNs were found (typically slightly > 1).
// backoff_factor:  Multiplier if infs/NaNs were found (typically 0.5).
// growth_interval:  Number of consecutive unskipped steps that must occur for current_scale to be multiplied by
//                   growth_factor.
// max_scale:  The maximum value the scale could grow.
// recover_threshold:  Allowing quickly recover the scaling factor when it is less or equal than this threshold.
// recover_growth_interval:  The growth_interval that will be used when the scaling factor is less or equal than
//                           the recover_threshold.
//
// Returns:
// current_scale
Tensor& _amp_update_scale_cuda_(Tensor& current_scale,
                                Tensor& growth_tracker,
                                const Tensor& found_inf,
                                double growth_factor,
                                double backoff_factor,
                                int64_t growth_interval,
                                double max_scale,
                                double recover_threshold,
                                int64_t recover_growth_interval)
{
  TORCH_CHECK(growth_tracker.is_cuda(), "growth_tracker must be a CUDA tensor.");
  TORCH_CHECK(current_scale.is_cuda(), "current_scale must be a CUDA tensor.");
  TORCH_CHECK(found_inf.is_cuda(), "found_inf must be a CUDA tensor.");
  TORCH_CHECK(growth_tracker.numel() == 1, "growth_tracker must be a 1-element tensor.");
  TORCH_CHECK(current_scale.numel() == 1, "current_scale must be a 1-element tensor.");
  TORCH_CHECK(found_inf.numel() == 1, "found_inf must be a 1-element tensor.");
  TORCH_CHECK(growth_tracker.scalar_type() == at::ScalarType::Int, "growth_tracker must be an int tensor.");
  TORCH_CHECK(current_scale.scalar_type() == at::ScalarType::Float, "current_scale must be a float tensor.");
  TORCH_CHECK(found_inf.scalar_type() == at::ScalarType::Float, "found_inf must be a float tensor.");

  amp_update_scale_cuda_kernel<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
    current_scale.data_ptr<float>(),
    growth_tracker.data_ptr<int>(),
    found_inf.data_ptr<float>(),
    growth_factor,
    backoff_factor,
    growth_interval,
    max_scale,
    recover_threshold,
    recover_growth_interval);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return current_scale;
}

TORCH_LIBRARY(modulus_ext, m) {
  m.def("_amp_update_scale_(Tensor(a!) self, Tensor(b!) growth_tracker, Tensor found_inf, float scale_growth_factor, float scale_backoff_factor, int growth_interval, float max_scale, float recover_threshold, int recover_growth_interval) -> Tensor(a!)");
}

TORCH_LIBRARY_IMPL(modulus_ext, CUDA, m) {
  m.impl("_amp_update_scale_", _amp_update_scale_cuda_);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {}
